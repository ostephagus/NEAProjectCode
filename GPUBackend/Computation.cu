#include "hip/hip_runtime.h"
#include "Computation.cuh"

constexpr BYTE SELF = 0b00010000;
constexpr BYTE NORTH = 0b00001000;
constexpr BYTE EAST = 0b00000100;
constexpr BYTE SOUTH = 0b00000010;
constexpr BYTE WEST = 0b00000001;

__global__ void ComputeRHS(PointerWithPitch<REAL> F, PointerWithPitch<REAL> G, PointerWithPitch<REAL> RHS, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY) {
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (rowNum > iMax) return; // Bounds checking
    if (colNum > jMax) return;
    
    *F_PITCHACCESS(RHS.ptr, RHS.pitch, rowNum, colNum) = (*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) * (1 / *timestep) * (((*F_PITCHACCESS(F.ptr, F.pitch, rowNum, colNum) - *F_PITCHACCESS(F.ptr, F.pitch, rowNum - 1, colNum)) / delX) + ((*F_PITCHACCESS(G.ptr, G.pitch, rowNum, colNum) - *F_PITCHACCESS(G.ptr, G.pitch, rowNum, colNum - 1)) / delY));
}