#include "hip/hip_runtime.h"
#include "Computation.cuh"
#include "DiscreteDerivatives.cuh"
#include <cmath>

constexpr BYTE SELF  = 0b00010000;
constexpr BYTE NORTH = 0b00001000;
constexpr BYTE EAST  = 0b00000100;
constexpr BYTE SELFSHIFT  = 4;
constexpr BYTE NORTHSHIFT = 3;
constexpr BYTE EASTSHIFT  = 2;


__global__ void ComputeFBoundary(PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> F, int iMax, int jMax) {
    int colNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (colNum > jMax) return;

    *F_PITCHACCESS(F.ptr, F.pitch, 0, colNum) = *F_PITCHACCESS(hVel.ptr, hVel.pitch, 0, colNum);
    *F_PITCHACCESS(F.ptr, F.pitch, iMax, colNum) = *F_PITCHACCESS(hVel.ptr, hVel.pitch, iMax, colNum);
}

__global__ void ComputeGBoundary(PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> G, int iMax, int jMax) {
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (rowNum > iMax) return;

    *F_PITCHACCESS(G.ptr, G.pitch, rowNum, 0) = *F_PITCHACCESS(vVel.ptr, vVel.pitch, rowNum, 0);
    *F_PITCHACCESS(G.ptr, G.pitch, rowNum, jMax) = *F_PITCHACCESS(vVel.ptr, vVel.pitch, rowNum, jMax);
}

__global__ void ComputeF(PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> F, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY, REAL xForce, REAL* gamma, REAL reynoldsNum) {
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (rowNum >= iMax) return;
    if (colNum > jMax) return;

    int selfBit = (*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) >> SELFSHIFT; // SELF bit of the cell's flag
    int eastBit = (*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & EAST) >> EASTSHIFT; // EAST bit of the cell's flag
    
    *F_PITCHACCESS(F.ptr, F.pitch, rowNum, colNum) =
        *F_PITCHACCESS(hVel.ptr, hVel.pitch, rowNum, colNum) * (selfBit | eastBit) // For boundary cells or fluid cells, add hVel
        + *timestep * (1 / reynoldsNum * (SecondPuPx(hVel, rowNum, colNum, delX) + SecondPuPy(hVel, rowNum, colNum, delY)) - PuSquaredPx(hVel, rowNum, colNum, delX, *gamma) - PuvPy(hVel, vVel, rowNum, colNum, delX, delY, *gamma) + xForce) * (selfBit & eastBit); // For fluid cells only, perform the computation. Obstacle cells without an eastern boundary are set to 0.
}


__global__ void ComputeG(PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> G, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY, REAL yForce, REAL* gamma, REAL reynoldsNum) {
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (rowNum > iMax) return;
    if (colNum >= jMax) return;

    int selfBit = (*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) >> SELFSHIFT;    // SELF bit of the cell's flag
    int northBit = (*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & NORTH) >> NORTHSHIFT; // NORTH bit of the cell's flag

    *F_PITCHACCESS(G.ptr, G.pitch, rowNum, colNum) =
        *F_PITCHACCESS(vVel.ptr, vVel.pitch, rowNum, colNum) * (selfBit | northBit) // For boundary cells or fluid cells, add vVel
        + *timestep * (1 / reynoldsNum * (SecondPvPx(vVel, rowNum, colNum, delX) + SecondPvPy(vVel, rowNum, colNum, delY)) - PuvPx(hVel, vVel, rowNum, colNum, delX, delY, *gamma) - PvSquaredPy(vVel, rowNum, colNum, delY, *gamma) + yForce) * (selfBit & northBit); // For fluid cells only, perform the computation. Obstacle cells without a northern boundary are set to 0.
}

hipError_t ComputeFG(hipStream_t* streams, dim3 threadsPerBlock, PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> F, PointerWithPitch<REAL> G, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY, REAL xForce, REAL yForce, REAL* gamma, REAL reynoldsNum) {
    dim3 numBlocksF((int)ceilf((float)(iMax - 1) / threadsPerBlock.x), (int)ceilf((float)jMax / threadsPerBlock.y));
    dim3 numBlocksG((int)ceilf((float)iMax / threadsPerBlock.x), (int)ceilf((float)(jMax - 1) / threadsPerBlock.y));

    int threadsPerBlockFlat = threadsPerBlock.x * threadsPerBlock.y;
    int numBlocksIMax = (int)ceilf((float)iMax / threadsPerBlockFlat);
    int numBlocksJMax = (int)ceilf((float)jMax / threadsPerBlockFlat);

    ComputeF<<<numBlocksF, threadsPerBlock, 0, streams[0]>>>(hVel, vVel, F, flags, iMax, jMax, timestep, delX, delY, xForce, gamma, reynoldsNum); // Launch the kernels in separate streams, to be concurrently executed if the GPU is able to.
    ComputeG<<<numBlocksG, threadsPerBlock, 0, streams[1]>>>(hVel, vVel, G, flags, iMax, jMax, timestep, delX, delY, yForce, gamma, reynoldsNum);

    ComputeFBoundary<<<numBlocksJMax, threadsPerBlockFlat, 0, streams[2]>>>(hVel, F, iMax, jMax);
    ComputeGBoundary<<<numBlocksIMax, threadsPerBlockFlat, 0, streams[3]>>>(vVel, G, iMax, jMax);

    return hipDeviceSynchronize();
}

__global__ void ComputeRHS(PointerWithPitch<REAL> F, PointerWithPitch<REAL> G, PointerWithPitch<REAL> RHS, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY) {
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (rowNum > iMax) return;
    if (colNum > jMax) return;
    
    *F_PITCHACCESS(RHS.ptr, RHS.pitch, rowNum, colNum) = 
        ((*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) >> SELFSHIFT) // Sets the entire expression to 0 if the cell is not fluid
        * (1 / *timestep) * (((*F_PITCHACCESS(F.ptr, F.pitch, rowNum, colNum) - *F_PITCHACCESS(F.ptr, F.pitch, rowNum - 1, colNum)) / delX) + ((*F_PITCHACCESS(G.ptr, G.pitch, rowNum, colNum) - *F_PITCHACCESS(G.ptr, G.pitch, rowNum, colNum - 1)) / delY));
}

__global__ void ComputeHVel(PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> F, PointerWithPitch<REAL> pressure, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX)
{
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (rowNum > iMax) return; // Bounds checking
    if (colNum > jMax) return;

    *F_PITCHACCESS(hVel.ptr, hVel.pitch, rowNum, colNum) =
        ((*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) >> SELFSHIFT) // Equal to 0 if the cell is not a fluid cell
        * ((*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & EAST) >> EASTSHIFT) // Equal to 0 if the cell has an obstacle cell next to it in +ve x direction (east)
        * (*F_PITCHACCESS(F.ptr, F.pitch, rowNum, colNum) - (*timestep / delX) * (*F_PITCHACCESS(pressure.ptr, pressure.pitch, rowNum + 1, colNum) - *F_PITCHACCESS(pressure.ptr, pressure.pitch, rowNum, colNum)));
}

__global__ void ComputeVVel(PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> G, PointerWithPitch<REAL> pressure, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delY)
{
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (rowNum > iMax) return; // Bounds checking
    if (colNum > jMax) return;

    *F_PITCHACCESS(vVel.ptr, vVel.pitch, rowNum, colNum) =
        ((*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) >> SELFSHIFT) // Equal to 0 if the cell is not a fluid cell
        * ((*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & NORTH) >> NORTHSHIFT) // Equal to 0 if the cell has an obstacle cell next to it in +ve y direction (north)
        * (*F_PITCHACCESS(G.ptr, G.pitch, rowNum, colNum) - (*timestep / delY) * (*F_PITCHACCESS(pressure.ptr, pressure.pitch, rowNum, colNum + 1) - *F_PITCHACCESS(pressure.ptr, pressure.pitch, rowNum, colNum)));
}

hipError_t ComputeVelocities(hipStream_t* streams, dim3 threadsPerBlock, PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> F, PointerWithPitch<REAL> G, PointerWithPitch<REAL> pressure, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY)
{
    dim3 numBlocks(iMax / threadsPerBlock.x, jMax / threadsPerBlock.y);
    ComputeHVel<<<numBlocks, threadsPerBlock, 0, streams[0]>>>(hVel, F, pressure, flags, iMax, jMax, timestep, delX); // Launch the kernels in separate streams, to be concurrently executed if the GPU is able to.
    ComputeVVel<<<numBlocks, threadsPerBlock, 0, streams[1]>>>(vVel, G, pressure, flags, iMax, jMax, timestep, delY);
    return hipDeviceSynchronize();
}

__global__ void ComputeStream(PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> streamFunction, int iMax, int jMax, REAL delY)
{
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (rowNum > iMax) return;

    *F_PITCHACCESS(streamFunction.ptr, streamFunction.pitch, rowNum, 0) = 0; // Stream function boundary condition
    for (int colNum = 1; colNum <= jMax; colNum++) {
        *F_PITCHACCESS(streamFunction.ptr, streamFunction.pitch, rowNum, colNum) = *F_PITCHACCESS(streamFunction.ptr, streamFunction.pitch, rowNum, colNum - 1) + *F_PITCHACCESS(hVel.ptr, hVel.pitch, rowNum, colNum) * delY;
    }
}
