#include "hip/hip_runtime.h"
#include "Computation.cuh"
#include "DiscreteDerivatives.cuh"
#include <cmath>

constexpr BYTE SELF  = 0b00010000;
constexpr BYTE NORTH = 0b00001000;
constexpr BYTE EAST  = 0b00000100;
constexpr BYTE SELFSHIFT  = 4;
constexpr BYTE NORTHSHIFT = 3;
constexpr BYTE EASTSHIFT  = 2;


__global__ void ComputeFBoundary(PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> F, int iMax, int jMax) {
    int colNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (colNum > jMax) return;

    *F_PITCHACCESS(F.ptr, F.pitch, 0, colNum) = *F_PITCHACCESS(hVel.ptr, hVel.pitch, 0, colNum);
    *F_PITCHACCESS(F.ptr, F.pitch, iMax, colNum) = *F_PITCHACCESS(hVel.ptr, hVel.pitch, iMax, colNum);
}

__global__ void ComputeGBoundary(PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> G, int iMax, int jMax) {
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (rowNum > iMax) return;

    *F_PITCHACCESS(G.ptr, G.pitch, rowNum, 0) = *F_PITCHACCESS(vVel.ptr, vVel.pitch, rowNum, 0);
    *F_PITCHACCESS(G.ptr, G.pitch, rowNum, jMax) = *F_PITCHACCESS(vVel.ptr, vVel.pitch, rowNum, jMax);
}

__global__ void ComputeF(PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> F, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY, REAL xForce, REAL* gamma, REAL reynoldsNum) {
    // Branchless plan:
    // if neither east nor self are set, set F to 0: east NOR self
    // if east or self but not both are set, set F to xVel: east XOR self
    // if both are set, set F to the equation: east AND self.
    // Compute east AND self. Multiply this by the larger computation.
    // Compute east OR self. Multiply this by xVel.
    // Add them.
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y;

    if (rowNum >= iMax) return;
    if (colNum >= jMax) return;

    int selfBit = (*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) >> SELFSHIFT;
    int eastBit = (*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & EAST) >> EASTSHIFT;

    *F_PITCHACCESS(F.ptr, F.pitch, rowNum, colNum) = 
        *F_PITCHACCESS(hVel.ptr, hVel.pitch, rowNum, colNum) * (selfBit | eastBit) // self OR east is either a fluid or boundary cell - these cells need xVel.
        + *timestep * (1 / reynoldsNum * (SecondPuPx(hVel, rowNum, colNum, delX) + SecondPuPy(hVel, rowNum, colNum, delY)) - PuSquaredPx(hVel, rowNum, colNum, delX, *gamma) - PuvPy(hVel, vVel, rowNum, colNum, delX, delY, *gamma) + xForce)
        * (selfBit & eastBit); // These are only fluid cells, perform the computation
}


__global__ void ComputeG(PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> G, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY, REAL yForce, REAL* gamma, REAL reynoldsNum) {
    // Branchless plan:
    // if neither east nor self are set, set F to 0: east NOR self
    // if east or self but not both are set, set F to xVel: east XOR self
    // if both are set, set F to the equation: east AND self.
    // Compute east AND self. Multiply this by the larger computation.
    // Compute east OR self. Multiply this by xVel.
    // Add them.
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y;

    if (rowNum >= iMax) return;
    if (colNum >= jMax) return;

    int selfBit = (*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) >> SELFSHIFT;
    int northBit = (*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & NORTH) >> NORTHSHIFT;

    *F_PITCHACCESS(G.ptr, G.pitch, rowNum, colNum) =
        *F_PITCHACCESS(vVel.ptr, vVel.pitch, rowNum, colNum) * (selfBit | northBit) // self OR east is either a fluid or boundary cell - these cells need xVel.
        + *timestep * (1 / reynoldsNum * (SecondPvPx(vVel, rowNum, colNum, delX) + SecondPvPy(vVel, rowNum, colNum, delY)) - PuvPx(hVel, vVel, rowNum, colNum, delX, delY, *gamma) - PvSquaredPy(vVel, rowNum, colNum, delY, *gamma) + yForce)
        * (selfBit & northBit); // These are only fluid cells, perform the computation
}

hipError_t ComputeFG(hipStream_t* streams, dim3 threadsPerBlock, PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> F, PointerWithPitch<REAL> G, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY, REAL xForce, REAL yForce, REAL* gamma, REAL reynoldsNum) {
    dim3 numBlocks((int)ceilf((float)(iMax - 1) / threadsPerBlock.x), (int)ceilf((float)(jMax - 1) / threadsPerBlock.y));

    int threadsPerBlockFlat = threadsPerBlock.x * threadsPerBlock.y;
    int numBlocksIMax = (int)ceilf((float)iMax / threadsPerBlockFlat);
    int numBlocksJMax = (int)ceilf((float)jMax / threadsPerBlockFlat);

    ComputeF<<<numBlocks, threadsPerBlock, 0, streams[0]>>>(hVel, vVel, F, flags, iMax, jMax, timestep, delX, delY, xForce, gamma, reynoldsNum); // Launch the kernels in separate streams, to be concurrently executed if the GPU is able to.
    ComputeG<<<numBlocks, threadsPerBlock, 0, streams[1]>>>(hVel, vVel, G, flags, iMax, jMax, timestep, delX, delY, yForce, gamma, reynoldsNum);

    ComputeFBoundary<<<numBlocksJMax, threadsPerBlockFlat, 0, streams[2]>>>(hVel, F, iMax, jMax);
    ComputeGBoundary<<<numBlocksIMax, threadsPerBlockFlat, 0, streams[3]>>>(vVel, G, iMax, jMax);

    return hipDeviceSynchronize();
}

__global__ void ComputeRHS(PointerWithPitch<REAL> F, PointerWithPitch<REAL> G, PointerWithPitch<REAL> RHS, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY) {
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (rowNum > iMax) return;
    if (colNum > jMax) return;
    
    *F_PITCHACCESS(RHS.ptr, RHS.pitch, rowNum, colNum) = 
        ((*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) >> SELFSHIFT) // Sets the entire expression to 0 if the cell is not fluid
        * (1 / *timestep) * (((*F_PITCHACCESS(F.ptr, F.pitch, rowNum, colNum) - *F_PITCHACCESS(F.ptr, F.pitch, rowNum - 1, colNum)) / delX) + ((*F_PITCHACCESS(G.ptr, G.pitch, rowNum, colNum) - *F_PITCHACCESS(G.ptr, G.pitch, rowNum, colNum - 1)) / delY));
}

__global__ void ComputeHVel(PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> F, PointerWithPitch<REAL> pressure, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX)
{
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (rowNum > iMax) return; // Bounds checking
    if (colNum > jMax) return;

    *F_PITCHACCESS(hVel.ptr, hVel.pitch, rowNum, colNum) =
        ((*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) >> SELFSHIFT) // Equal to 0 if the cell is not a fluid cell
        * ((*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & EAST) >> EASTSHIFT) // Equal to 0 if the cell has an obstacle cell next to it in +ve x direction (east)
        * (*F_PITCHACCESS(F.ptr, F.pitch, rowNum, colNum) - (*timestep / delX) * (*F_PITCHACCESS(pressure.ptr, pressure.pitch, rowNum + 1, colNum) - *F_PITCHACCESS(pressure.ptr, pressure.pitch, rowNum, colNum)));
}

__global__ void ComputeVVel(PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> G, PointerWithPitch<REAL> pressure, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delY)
{
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (rowNum > iMax) return; // Bounds checking
    if (colNum > jMax) return;

    *F_PITCHACCESS(vVel.ptr, vVel.pitch, rowNum, colNum) =
        ((*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) >> SELFSHIFT) // Equal to 0 if the cell is not a fluid cell
        * ((*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & NORTH) >> NORTHSHIFT) // Equal to 0 if the cell has an obstacle cell next to it in +ve y direction (north)
        * (*F_PITCHACCESS(G.ptr, G.pitch, rowNum, colNum) - (*timestep / delY) * (*F_PITCHACCESS(pressure.ptr, pressure.pitch, rowNum, colNum + 1) - *F_PITCHACCESS(pressure.ptr, pressure.pitch, rowNum, colNum)));
}

hipError_t ComputeVelocities(hipStream_t* streams, dim3 threadsPerBlock, PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> F, PointerWithPitch<REAL> G, PointerWithPitch<REAL> pressure, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY)
{
    dim3 numBlocks(iMax / threadsPerBlock.x, jMax / threadsPerBlock.y);
    ComputeHVel << <numBlocks, threadsPerBlock, 0, streams[0] >> > (hVel, F, pressure, flags, iMax, jMax, timestep, delX); // Launch the kernels in separate streams, to be concurrently executed if the GPU is able to.
    ComputeVVel << <numBlocks, threadsPerBlock, 0, streams[1] >> > (vVel, G, pressure, flags, iMax, jMax, timestep, delY);
    return hipDeviceSynchronize();
}

__global__ void ComputeStream(PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> streamFunction, int iMax, int jMax, REAL delY)
{
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (rowNum > iMax) return;

    *F_PITCHACCESS(streamFunction.ptr, streamFunction.pitch, rowNum, 0) = 0; // Stream function boundary condition
    for (int colNum = 1; colNum <= jMax; colNum++) {
        *F_PITCHACCESS(streamFunction.ptr, streamFunction.pitch, rowNum, colNum) = *F_PITCHACCESS(streamFunction.ptr, streamFunction.pitch, rowNum, colNum - 1) + *F_PITCHACCESS(hVel.ptr, hVel.pitch, rowNum, colNum) * delY;
    }
}
