#include "hip/hip_runtime.h"
#include "Computation.cuh"
#include "DiscreteDerivatives.cuh"
#include <cmath>

constexpr BYTE SELF  = 0b00010000;
constexpr BYTE NORTH = 0b00001000;
constexpr BYTE EAST  = 0b00000100;
constexpr BYTE SELFSHIFT  = 4;
constexpr BYTE NORTHSHIFT = 3;
constexpr BYTE EASTSHIFT  = 2;


__device__ void GroupMax(cg::thread_group group, volatile REAL* sharedArray) {
    int index = group.thread_rank();
    REAL val = sharedArray[index];
    for (int indexThreshold = group.size() / 2; indexThreshold > 0; indexThreshold /= 2) {
        if (index < indexThreshold) { // Halve the number of threads each iteration
            val = fmaxf(val, sharedArray[index + indexThreshold]); // Get the max of the thread's own value and the one at index + indexThreshold
            sharedArray[index] = val; // Store the max into the shared array at the current index
        }
        group.sync();
    }
}

__global__ void ComputePartialMaxes(REAL* partialMaxes, PointerWithPitch<REAL> field, int yLength) {
    cg::thread_block threadBlock = cg::this_thread_block();
    REAL* colBase = (REAL*)((char*)field.ptr + blockIdx.x * field.pitch);

    // Perform copy to shared memory.
    // Put a 0 in shared if current index is greater than yLength (this catches index in pitch padding, or index > size of a row)
    extern __shared__ REAL sharedArray[];

    if (threadIdx.x < yLength) { // the index of the thread is greater than the length of a column.
        sharedArray[threadIdx.x] = *(colBase + threadIdx.x);
    }
    else {
        sharedArray[threadIdx.x] = (REAL)0;
    }
    threadBlock.sync();

    GroupMax(threadBlock, sharedArray);

    if (threadIdx.x == 0) { // If the thread is the 0th in the block, store its result to global memory.
        partialMaxes[blockIdx.x] = sharedArray[0];
    }
}

__global__ void ComputeFinalMax(REAL* max, REAL* partialMaxes, int xLength)
{
    cg::thread_block threadBlock = cg::this_thread_block();

    extern __shared__ REAL sharedMem[];

    // Copy to shared memory again
    if (threadIdx.x < xLength) {
        sharedMem[threadIdx.x] = partialMaxes[threadIdx.x];
    }
    else {
        sharedMem[threadIdx.x] = (REAL)0;
    }
    threadBlock.sync();

    GroupMax(threadBlock, sharedMem);
    if (threadIdx.x == 0) { // Thread 0 stores the final element.
        *max = sharedMem[0];
    }
}

hipError_t FieldMax(REAL* max, hipStream_t streamToUse, PointerWithPitch<REAL> field, int xLength, int yLength) {
    hipError_t retVal;

    REAL* partialMaxes;
    retVal = hipMalloc(&partialMaxes, xLength * sizeof(REAL));
    if (retVal != hipSuccess) { // Return if there was an error in allocation
        return retVal;
    }

    // Run the GPU kernel:
    ComputePartialMaxes KERNEL_ARGS4(xLength, (unsigned int)field.pitch / sizeof(REAL), field.pitch, streamToUse) (partialMaxes, field, yLength); // 1 block per row. Number of threads is equal to column pitch, and each thread has 1 REAL worth of shared memory.
    retVal = hipStreamSynchronize(streamToUse);
    if (retVal != hipSuccess) { // Skip the rest of the computation if there was an error
        goto free;
    }

    ComputeFinalMax KERNEL_ARGS4(1, xLength, xLength * sizeof(REAL), streamToUse) (max, partialMaxes, xLength); // 1 block to process all of the partial maxes, number of threads equal to number of partial maxes (xLength is also this)
    retVal = hipStreamSynchronize(streamToUse);


free:
    hipFree(partialMaxes);
    return retVal;
}

__global__ void FinishComputeGamma(REAL* gamma, REAL* hVelMax, REAL* vVelMax, REAL* timestep, REAL delX, REAL delY) {
    REAL horizontalComponent = *hVelMax * (*timestep / delX);
    REAL verticalComponent = *vVelMax * (*timestep / delY);

    if (horizontalComponent > verticalComponent) {
        *gamma = horizontalComponent;
    }
    else {
        *gamma = verticalComponent;
    }
}

hipError_t ComputeGamma(REAL* gamma, hipStream_t* streams, int threadsPerBlock, PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> vVel, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY) {
    hipError_t retVal;
    REAL* hVelMax;
    retVal = hipMalloc(&hVelMax, sizeof(REAL));
    if (retVal != hipSuccess) goto free;

    REAL* vVelMax;
    retVal = hipMalloc(&vVelMax, sizeof(REAL));
    if (retVal != hipSuccess) goto free;

    FieldMax(hVelMax, streams[0], hVel, iMax + 2, jMax + 2);

    retVal = hipStreamSynchronize(streams[0]);
    if (retVal != hipSuccess) goto free;

    FieldMax(vVelMax, streams[1], vVel, iMax + 2, jMax + 2);

    retVal = hipStreamSynchronize(streams[1]);
    if (retVal != hipSuccess) goto free;

    FinishComputeGamma KERNEL_ARGS4(1, 1, 0, streams[0]) (gamma, hVelMax, vVelMax, timestep, delX, delY);

    free:
    hipFree(hVelMax);
    hipFree(vVelMax);
    return retVal;
}

__global__ void FinishComputeTimestep(REAL* timestep, REAL* hVelMax, REAL* vVelMax, REAL delX, REAL delY, REAL reynoldsNo, REAL safetyFactor)
{
    REAL inverseSquareRestriction = (REAL)0.5 * reynoldsNo * (1 / square(delX) + 1 / square(delY));
    REAL xTravelRestriction = delX / *hVelMax;
    REAL yTravelRestriction = delY / *vVelMax;

    REAL smallestRestriction = inverseSquareRestriction; // Choose the smallest restriction
    if (xTravelRestriction < smallestRestriction) {
        smallestRestriction = xTravelRestriction;
    }
    if (yTravelRestriction < smallestRestriction) {
        smallestRestriction = yTravelRestriction;
    }
    *timestep = safetyFactor * smallestRestriction;
}

hipError_t ComputeTimestep(REAL* timestep, hipStream_t* streams, PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> vVel, int iMax, int jMax, REAL delX, REAL delY, REAL reynoldsNo, REAL safetyFactor)
{
    hipError_t retVal;
    REAL* hVelMax;
    retVal = hipMalloc(&hVelMax, sizeof(REAL));
    if (retVal != hipSuccess) goto free;

    REAL* vVelMax;
    retVal = hipMalloc(&vVelMax, sizeof(REAL));
    if (retVal != hipSuccess) goto free;

    FieldMax(hVelMax, streams[0], hVel, iMax + 2, jMax + 2);

    retVal = hipStreamSynchronize(streams[0]);
    if (retVal != hipSuccess) goto free;

    FieldMax(vVelMax, streams[1], vVel, iMax + 2, jMax + 2);

    retVal = hipStreamSynchronize(streams[1]);
    if (retVal != hipSuccess) goto free;

    FinishComputeTimestep KERNEL_ARGS4(1, 1, 0, streams[0]) (timestep, hVelMax, vVelMax, delX, delY, reynoldsNo, safetyFactor);

free:
    hipFree(hVelMax);
    hipFree(vVelMax);
    return retVal;
}

__global__ void ComputeFBoundary(PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> F, int iMax, int jMax) {
    int colNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (colNum > jMax) return;

    *F_PITCHACCESS(F.ptr, F.pitch, 0, colNum) = *F_PITCHACCESS(hVel.ptr, hVel.pitch, 0, colNum);
    *F_PITCHACCESS(F.ptr, F.pitch, iMax, colNum) = *F_PITCHACCESS(hVel.ptr, hVel.pitch, iMax, colNum);
}

__global__ void ComputeGBoundary(PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> G, int iMax, int jMax) {
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (rowNum > iMax) return;

    *F_PITCHACCESS(G.ptr, G.pitch, rowNum, 0) = *F_PITCHACCESS(vVel.ptr, vVel.pitch, rowNum, 0);
    *F_PITCHACCESS(G.ptr, G.pitch, rowNum, jMax) = *F_PITCHACCESS(vVel.ptr, vVel.pitch, rowNum, jMax);
}

__global__ void ComputeF(PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> F, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY, REAL xForce, REAL* gamma, REAL reynoldsNum) {
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (rowNum >= iMax) return;
    if (colNum > jMax) return;

    int selfBit = (*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) >> SELFSHIFT; // SELF bit of the cell's flag
    int eastBit = (*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & EAST) >> EASTSHIFT; // EAST bit of the cell's flag
    
    *F_PITCHACCESS(F.ptr, F.pitch, rowNum, colNum) =
        *F_PITCHACCESS(hVel.ptr, hVel.pitch, rowNum, colNum) * (selfBit | eastBit) // For boundary cells or fluid cells, add hVel
        + *timestep * (1 / reynoldsNum * (SecondPuPx(hVel, rowNum, colNum, delX) + SecondPuPy(hVel, rowNum, colNum, delY)) - PuSquaredPx(hVel, rowNum, colNum, delX, *gamma) - PuvPy(hVel, vVel, rowNum, colNum, delX, delY, *gamma) + xForce) * (selfBit & eastBit); // For fluid cells only, perform the computation. Obstacle cells without an eastern boundary are set to 0.
}


__global__ void ComputeG(PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> G, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY, REAL yForce, REAL* gamma, REAL reynoldsNum) {
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (rowNum > iMax) return;
    if (colNum >= jMax) return;

    int selfBit = (*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) >> SELFSHIFT;    // SELF bit of the cell's flag
    int northBit = (*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & NORTH) >> NORTHSHIFT; // NORTH bit of the cell's flag

    *F_PITCHACCESS(G.ptr, G.pitch, rowNum, colNum) =
        *F_PITCHACCESS(vVel.ptr, vVel.pitch, rowNum, colNum) * (selfBit | northBit) // For boundary cells or fluid cells, add vVel
        + *timestep * (1 / reynoldsNum * (SecondPvPx(vVel, rowNum, colNum, delX) + SecondPvPy(vVel, rowNum, colNum, delY)) - PuvPx(hVel, vVel, rowNum, colNum, delX, delY, *gamma) - PvSquaredPy(vVel, rowNum, colNum, delY, *gamma) + yForce) * (selfBit & northBit); // For fluid cells only, perform the computation. Obstacle cells without a northern boundary are set to 0.
}

hipError_t ComputeFG(hipStream_t* streams, dim3 threadsPerBlock, PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> F, PointerWithPitch<REAL> G, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY, REAL xForce, REAL yForce, REAL* gamma, REAL reynoldsNum) {
    dim3 numBlocksF(INT_DIVIDE_ROUND_UP(iMax - 1, threadsPerBlock.x), INT_DIVIDE_ROUND_UP(jMax, threadsPerBlock.y));
    dim3 numBlocksG(INT_DIVIDE_ROUND_UP(iMax, threadsPerBlock.x), INT_DIVIDE_ROUND_UP(jMax - 1, threadsPerBlock.y));

    int threadsPerBlockFlat = threadsPerBlock.x * threadsPerBlock.y;
    int numBlocksIMax = INT_DIVIDE_ROUND_UP(iMax, threadsPerBlockFlat);
    int numBlocksJMax = INT_DIVIDE_ROUND_UP(jMax, threadsPerBlockFlat);

    ComputeF KERNEL_ARGS4(numBlocksF, threadsPerBlock, 0, streams[0]) (hVel, vVel, F, flags, iMax, jMax, timestep, delX, delY, xForce, gamma, reynoldsNum); // Launch the kernels in separate streams, to be concurrently executed if the GPU is able to.
    ComputeG KERNEL_ARGS4(numBlocksG, threadsPerBlock, 0, streams[1]) (hVel, vVel, G, flags, iMax, jMax, timestep, delX, delY, yForce, gamma, reynoldsNum);

    ComputeFBoundary KERNEL_ARGS4(numBlocksJMax, threadsPerBlockFlat, 0, streams[2]) (hVel, F, iMax, jMax);
    ComputeGBoundary KERNEL_ARGS4(numBlocksIMax, threadsPerBlockFlat, 0, streams[3]) (vVel, G, iMax, jMax);

    return hipDeviceSynchronize();
}

__global__ void ComputeRHS(PointerWithPitch<REAL> F, PointerWithPitch<REAL> G, PointerWithPitch<REAL> RHS, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY) {
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (rowNum > iMax) return;
    if (colNum > jMax) return;
    
    *F_PITCHACCESS(RHS.ptr, RHS.pitch, rowNum, colNum) = 
        ((*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) >> SELFSHIFT) // Sets the entire expression to 0 if the cell is not fluid
        * (1 / *timestep) * (((*F_PITCHACCESS(F.ptr, F.pitch, rowNum, colNum) - *F_PITCHACCESS(F.ptr, F.pitch, rowNum - 1, colNum)) / delX) + ((*F_PITCHACCESS(G.ptr, G.pitch, rowNum, colNum) - *F_PITCHACCESS(G.ptr, G.pitch, rowNum, colNum - 1)) / delY));
}

__global__ void ComputeHVel(PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> F, PointerWithPitch<REAL> pressure, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX)
{
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (rowNum > iMax) return; // Bounds checking
    if (colNum > jMax) return;

    *F_PITCHACCESS(hVel.ptr, hVel.pitch, rowNum, colNum) =
        ((*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) >> SELFSHIFT) // Equal to 0 if the cell is not a fluid cell
        * ((*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & EAST) >> EASTSHIFT) // Equal to 0 if the cell has an obstacle cell next to it in +ve x direction (east)
        * (*F_PITCHACCESS(F.ptr, F.pitch, rowNum, colNum) - (*timestep / delX) * (*F_PITCHACCESS(pressure.ptr, pressure.pitch, rowNum + 1, colNum) - *F_PITCHACCESS(pressure.ptr, pressure.pitch, rowNum, colNum)));
}

__global__ void ComputeVVel(PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> G, PointerWithPitch<REAL> pressure, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delY)
{
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int colNum = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (rowNum > iMax) return; // Bounds checking
    if (colNum > jMax) return;

    *F_PITCHACCESS(vVel.ptr, vVel.pitch, rowNum, colNum) =
        ((*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & SELF) >> SELFSHIFT) // Equal to 0 if the cell is not a fluid cell
        * ((*B_PITCHACCESS(flags.ptr, flags.pitch, rowNum, colNum) & NORTH) >> NORTHSHIFT) // Equal to 0 if the cell has an obstacle cell next to it in +ve y direction (north)
        * (*F_PITCHACCESS(G.ptr, G.pitch, rowNum, colNum) - (*timestep / delY) * (*F_PITCHACCESS(pressure.ptr, pressure.pitch, rowNum, colNum + 1) - *F_PITCHACCESS(pressure.ptr, pressure.pitch, rowNum, colNum)));
}

hipError_t ComputeVelocities(hipStream_t* streams, dim3 threadsPerBlock, PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> vVel, PointerWithPitch<REAL> F, PointerWithPitch<REAL> G, PointerWithPitch<REAL> pressure, PointerWithPitch<BYTE> flags, int iMax, int jMax, REAL* timestep, REAL delX, REAL delY)
{
    dim3 numBlocks(INT_DIVIDE_ROUND_UP(iMax, threadsPerBlock.x), INT_DIVIDE_ROUND_UP(jMax, threadsPerBlock.y));
    ComputeHVel KERNEL_ARGS4(numBlocks, threadsPerBlock, 0, streams[0]) (hVel, F, pressure, flags, iMax, jMax, timestep, delX); // Launch the kernels in separate streams, to be concurrently executed if the GPU is able to.
    ComputeVVel KERNEL_ARGS4(numBlocks, threadsPerBlock, 0, streams[1]) (vVel, G, pressure, flags, iMax, jMax, timestep, delY);
    return hipDeviceSynchronize();
}

__global__ void ComputeStream(PointerWithPitch<REAL> hVel, PointerWithPitch<REAL> streamFunction, int iMax, int jMax, REAL delY)
{
    int rowNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (rowNum > iMax) return;

    *F_PITCHACCESS(streamFunction.ptr, streamFunction.pitch, rowNum, 0) = 0; // Stream function boundary condition
    for (int colNum = 1; colNum <= jMax; colNum++) {
        *F_PITCHACCESS(streamFunction.ptr, streamFunction.pitch, rowNum, colNum) = *F_PITCHACCESS(streamFunction.ptr, streamFunction.pitch, rowNum, colNum - 1) + *F_PITCHACCESS(hVel.ptr, hVel.pitch, rowNum, colNum) * delY;
    }
}
