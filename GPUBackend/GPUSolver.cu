#include "hip/hip_runtime.h"
#include "GPUSolver.cuh"
#include "Init.h"
#include "Boundary.cuh"
#include "Computation.cuh"
#include "math.h"

GPUSolver::GPUSolver(SimulationParameters parameters, int iMax, int jMax) : Solver(parameters, iMax, jMax) {
    hVel = pointerWithPitch<REAL>();
    hipMallocPitch(&hVel.ptr, &hVel.pitch, (jMax + 2) * sizeof(REAL), iMax + 2);

    vVel = pointerWithPitch<REAL>();
    hipMallocPitch(&vVel.ptr, &vVel.pitch, (jMax + 2) * sizeof(REAL), iMax + 2);

    pressure = pointerWithPitch<REAL>();
    hipMallocPitch(&pressure.ptr, &pressure.pitch, (jMax + 2) * sizeof(REAL), iMax + 2);

    RHS = pointerWithPitch<REAL>();
    hipMallocPitch(&RHS.ptr, &RHS.pitch, (jMax + 2) * sizeof(REAL), iMax + 2);

    F = pointerWithPitch<REAL>();
    hipMallocPitch(&F.ptr, &F.pitch, (jMax + 2) * sizeof(REAL), iMax + 2);

    G = pointerWithPitch<REAL>();
    hipMallocPitch(&G.ptr, &G.pitch, (jMax + 2) * sizeof(REAL), iMax + 2);

    streamFunction = pointerWithPitch<REAL>();
    hipMallocPitch(&streamFunction.ptr, &streamFunction.pitch, (jMax + 2) * sizeof(REAL), iMax + 2);

    devFlags = pointerWithPitch<BYTE>();
    hipMallocPitch(&devFlags.ptr, &devFlags.pitch, (jMax + 2) * sizeof(BYTE), iMax + 2);
        
    hostFlags = FlagMatrixMAlloc(iMax + 2, jMax + 2);
    obstacles = nullptr;
}

GPUSolver::~GPUSolver() {
    hipFree(hVel.ptr);
    hipFree(vVel.ptr);
    hipFree(pressure.ptr);
    hipFree(RHS.ptr);
    hipFree(F.ptr);
    hipFree(G.ptr);
    hipFree(streamFunction.ptr);
    hipFree(devFlags.ptr);
    FreeMatrix(hostFlags, iMax + 2);
    FreeMatrix(obstacles, iMax + 2);
}

void GPUSolver::SetBlockDimensions()
{
    // The below code takes the square root of the number of threads, but if the number of threads per block is not a square it takes the powers of 2 either side of the square root.
    // For example, a maxThreadsPerBlock of 1024 would mean threadsPerBlock becomes 32 and 32, but a maxThreadsPerBlock of 512 would mean threadsPerBlock would become 32 and 16
    int maxThreadsPerBlock = deviceProperties.maxThreadsPerBlock;
    int log2ThreadsPerBlock = (int)ceilf(log2f((float)maxThreadsPerBlock)); // Threads per block should be a power of 2, but ceil just in case
    int log2XThreadsPerBlock = (int)ceilf((float)log2ThreadsPerBlock / 2.0f); // Divide by 2, if log2(threadsPerBlock) was odd, ceil
    int log2YThreadsPerBlock = (int)floorf((float)log2ThreadsPerBlock / 2.0f); // As above, but floor for smaller one
    int xThreadsPerBlock = (int)powf((float)log2XThreadsPerBlock, 2); // Now exponentiate to get the actual number of threads
    int yThreadsPerBlock = (int)powf((float)log2YThreadsPerBlock, 2);
    threadsPerBlock = dim3(xThreadsPerBlock, yThreadsPerBlock);

    int blocksForIMax = (int)ceilf((float)iMax / threadsPerBlock.x);
    int blocksForJMax = (int)ceilf((float)jMax / threadsPerBlock.y);
    numBlocks = dim3(blocksForIMax, blocksForJMax);
}

void GPUSolver::CreatePointerArray(REAL** ptrArray, REAL* valueArray, int stride, int count)
{
    for (int i = 0; i < count; i++) {
        ptrArray[i] = valueArray + i * stride; // Set the pointer at the certain index to however far along the flattened array the next row is
    }
}

bool** GPUSolver::GetObstacles() {
    if (obstacles == nullptr) {
        obstacles = ObstacleMatrixMAlloc(iMax + 2, jMax + 2);
    }
    return obstacles;
}

void GPUSolver::ProcessObstacles() {
    SetFlags(obstacles, hostFlags, iMax + 2, jMax + 2); // SetFlags is done on the CPU


}

void GPUSolver::PerformSetup() {
    hipGetDeviceProperties(&deviceProperties, 0);

    SetBlockDimensions();


}

void GPUSolver::Timestep(REAL& simulationTime) {
    SetBoundaryConditions(streams, deviceProperties.maxThreadsPerBlock, hVel, vVel, devFlags, coordinates, coordinatesLength, iMax, jMax, parameters.inflowVelocity, parameters.surfaceFrictionalPermissibility);

    //REAL timestep;
    // Compute timestep
    //simulationTime += timestep;

    // Compute gamma
    // Compute F and G
    
    ComputeRHS<<<numBlocks, threadsPerBlock>>>(F, G, RHS, iMax, jMax, timestep, delX, delY);

    // Compute pressure Poisson
    // Compute velocities
    // Compute stream function
}

